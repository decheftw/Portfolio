//////////////////////////////////////////////////////////////////////////
////This is the code implementation for Hanon finger exercise -- memory
////Dartmouth COSC89.25/189.03, GPU Programming and High-Performance Computing
////Team DNS: Gao Chen, Nicolas Flores, Shikhar Sinha
//////////////////////////////////////////////////////////////////////////


#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>
#include <iostream>
#include <fstream>
using namespace std;

//////////////////////////////////////////////////////////////////////////
////TODO 0: Please replace the following strings with your team name and author names
////Note: Please do not use space in the string, use "_" instead
//////////////////////////////////////////////////////////////////////////

namespace name
{
	std::string team="dns";
	std::string author_1="Gao_Chen";
	std::string author_2="Nicolas_Flores";
	std::string author_3="Shikhar_Sinha";	////optional
};

ofstream out;

//////////////////////////////////////////////////////////////////////////
////Hanon finger exercise for memory manipulations
////In this exercise you will practice the use of a set of CUDA memory APIs, 
////  including cudaMalloc, cudaFree, cudaMemcpy, cudaMemcpyFrom(To)Symbol, and cudaGetSymbolAddress
//// For the API manual, please visit: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__MEMORY.html#group__CUDART__MEMORY
//// Please also read the in-class sample code on GitLab before you start this exercise, including lec5_global_memory.cu and lec6_local_memory.cu

const int a_host[8]={1,2,3,4,5,6,7,8};								////a_host is an array on host
__device__ const int b_dev[8]={101,102,103,104,105,106,107,108};	////b_dev is an array on device

////Hanon Exercise 12: practice cudaMalloc, cudaMemcpy, and cudaFree
////Expected output: result_host={2,3,4,5,6,7,8,9}
////Process: copy a_host from host to a dynamic array that you created on device (not b_dev declared above!), add each of its elements by 1, and then store the results in result_host
////Hint:
////0) allocate a dynamic array on device with the same size as a_host;
////1) copy a_host from host to device;
////2) write a kernel function to carry out the incremental operation on device;
////3) copy the calculated results back from device to result_host (on host)
////4) free the array on device

/*TODO: Your kernel function starts*/
__global__ void copy_and_add(int *arr)
{
    arr[threadIdx.x] += 1;
}

/*TODO: Your kernel function ends*/

__host__ void Hanon_Exercise_12()
{
	int result_host[8]={0};
	
    /*TODO: Your implementation starts*/
    int *a_dev;
    hipMalloc((void**)&a_dev, 8*sizeof(int));
    hipMemcpy(a_dev,a_host,8*sizeof(int),hipMemcpyHostToDevice);
    copy_and_add<<<1,8, 8*sizeof(int)>>>(a_dev);
	hipMemcpy(result_host,a_dev, 8*sizeof(int), hipMemcpyDeviceToHost);
	hipFree((void*)a_dev);

	/*TODO: Your implementation ends*/

	cout<<"Hanon exercise 12:\n";
	for(int i=0;i<8;i++)cout<<result_host[i]<<", ";cout<<endl;
	out<<"Hanon exercise 12:\n";
	for(int i=0;i<8;i++)out<<result_host[i]<<", ";out<<endl;
}

////Hanon Exercise 13: practice cudaMemcpyFromSymbol
////Expected output: result_host={101,102,103,104,105,106,107,108}
////Process: copy b_dev (the static CUDA device array declared in line 35) to result_host by using cudaMemcpyFromSymbol. 
////Hint: b_dev is in static (stack) memory, so you cannot use cudaMemcpy to manipulate it!
__host__ void Hanon_Exercise_13()
{
	vector<int> result_host(8,0);
	
    /*TODO: Your implementation starts*/
    // cudaGetSymbolAddress((void**)&b_dev, b_dev);
    // cudaMemcpy((void*)b_dev, (void*)&result_host[0], 8 * sizeof(int), cudaMemcpyDeviceToHost);

    hipMemcpyFromSymbol(&result_host[0],HIP_SYMBOL(b_dev),8*sizeof(int));
	/*TODO: Your implementation ends*/

	cout<<"Hanon exercise 13:\n";
	for(int i=0;i<8;i++)cout<<result_host[i]<<", ";cout<<endl;
	out<<"Hanon exercise 13:\n";
	for(int i=0;i<8;i++)out<<result_host[i]<<", ";out<<endl;
}

////Hanon Exercise 14: practice manipulating dynamic and static memories together
////Expected output: result_host={101+1,102+2,103+3,104+4,105+5,106+6,107+7,108+8}
////Process: calculate a_host+b_dev (element-wise sum) on device and store the results in result_host
////Hint:
////1) transferring a_host from host to device;
////2) write a kernel function to carry out the element-wise sum for arrays a_host and b_dev
////3) transfer the results from device to result_host (on host)

/*TODO: Your kernel function starts*/
__global__ void elem_sum(int *arr)
{
    arr[threadIdx.x] += b_dev[threadIdx.x];
}
/*TODO: Your kernel function ends*/

__host__ void Hanon_Exercise_14()
{
	int result_host[8]={0};
	
    /*TODO: Your host function implementation starts*/
    int *a_dev;
    hipMalloc((void**)&a_dev, 8*sizeof(int));
    hipMemcpy(a_dev,a_host,8*sizeof(int),hipMemcpyHostToDevice);
    elem_sum<<<1,8, 8*sizeof(int)>>>(a_dev);
	hipMemcpy(result_host,a_dev, 8*sizeof(int), hipMemcpyDeviceToHost);
	hipFree((void*)a_dev);

	/*TODO: Your host function implementation ends*/

	cout<<"Hanon exercise 14:\n";
	for(int i=0;i<8;i++)cout<<result_host[i]<<", ";cout<<endl;
	out<<"Hanon exercise 14:\n";
	for(int i=0;i<8;i++)out<<result_host[i]<<", ";out<<endl;
}

////Hanon Exercise 15: practice using shared memory
////Expected output: result_host={1*0+101,2*2+102,3*4+103,4*6+104,5*8+105,6*10+106,7*12+107,8*14+108}
////Process: calculate a_host*s+b_dev and store results in result_host. Here s is an array initialized in shared memory of the kernel function (line 111-113)
////Hint: You need to modify the arguments and the implementation of the function Calculate_Array_With_Shared() to pass in your array(s) and perform calculations 

__global__ void Calculate_Array_With_Shared(int *a)	/*TODO: modify the arguments of the kernel function*/
{
	__shared__ int s[8];
	s[threadIdx.x]=2*threadIdx.x;
	__syncthreads();

    /*TODO: Your kernel implementation starts*/
    a[threadIdx.x] = a[threadIdx.x]*s[threadIdx.x]+b_dev[threadIdx.x];
	/*TODO: Your kernel implementation ends*/
}

__host__ void Hanon_Exercise_15()
{	
	/*TODO: Your host function implementation starts*/
	/*TODO: Your host function implementation ends*/

	int result_host[8]={0};	
    int *a_dev;
    hipMalloc((void**)&a_dev, 8*sizeof(int));
    hipMemcpy(a_dev,a_host,8*sizeof(int),hipMemcpyHostToDevice);
    Calculate_Array_With_Shared<<<1,8, 8*sizeof(int)>>>(a_dev);
	hipMemcpy(result_host,a_dev, 8*sizeof(int), hipMemcpyDeviceToHost);
	hipFree((void*)a_dev);


	cout<<"Hanon exercise 15:\n";
	for(int i=0;i<8;i++)cout<<result_host[i]<<", ";cout<<endl;
	out<<"Hanon exercise 15:\n";
	for(int i=0;i<8;i++)out<<result_host[i]<<", ";out<<endl;
}

////Hanon Exercise 16: practice cudaGetSymbolAddress
////Expected output: result_host={101*16+1,102*16+1,103*16+1,...,108*16+1}
////Process: apply the following kernel function Manipulate_Array() onto b_dev and store the results in result_host
////*WITHOUT* modifying the implementation in Manipulate_Array() (call it as a blackbox)
////Hint: b_dev is a static array on GPU, you need to get its dynamic pointer by calling cudaGetSymbolAddress, and then send this pointer into the kernel function to update its values

////Note: You are not allowed to modify the implementation in this function!
__global__ void Manipulate_Array(int* array)
{
	array[threadIdx.x]*=16;
	array[threadIdx.x]+=1;
}

__host__ void Hanon_Exercise_16()
{
	/*TODO: Your host function implementation starts*/

	int *a_dev;
	int result_host[8]={0};	
    //cudaMalloc((void**)&a_dev, 8*sizeof(int));
	hipGetSymbolAddress((void**)&a_dev, HIP_SYMBOL(b_dev));
	Manipulate_Array<<<1, 8, 8*sizeof(int)>>>(a_dev);
	hipMemcpy(result_host,a_dev, 8*sizeof(int), hipMemcpyDeviceToHost);
	/*TODO: Your host function implementation ends*/

	// int result_host[8]={0};	

	cout<<"Hanon exercise 16:\n";
	for(int i=0;i<8;i++)cout<<result_host[i]<<", ";cout<<endl;
	out<<"Hanon exercise 16:\n";
	for(int i=0;i<8;i++)out<<result_host[i]<<", ";out<<endl;
}

////Hanon Exercise 17: practice using shared memory with multiple array types
////Expected output: array_int={208,206,204,202}, array_float={8.,6.,4.,2.}, 
//// i.e., reverse the order of the int array, multiply each element by 2, and copy its values to the float array (by type conversion), 
//// and reverse the order of the float array, multiply each element by 2, and copy its values to the int array (by type conversion)
//// You need to implement this process by using a piece of shared memory holding both two arrays
////Hint: read the sample code we went through in class on Thursday, and mimic its steps as
////1. Initialize two array pointers with the types of int and float to different addresses of the shared memory
////2. Copy the values from array_int and array_float to the proper elements in shared memory
////3. synchronize threads
////4. Copy the values with the proper order and rescaling factor from each array in shared memory to global memory (array_int and array_float)

__global__ void Reverse_And_Multiply_Two_Arrays_With_Extern_Shared(int* array_int,const size_t array_int_size,float* array_float,const size_t array_float_size)
{
	extern __shared__ int shared_mem[];
	// int* ai= /*something you need to specify on shared memory*/0;
	// float* af= /*something you need to specify on shared memory*/0;
	int s = array_int_size + array_float_size;
	/*Your implementation*/
	shared_mem[threadIdx.x] = array_int[threadIdx.x];
	shared_mem[threadIdx.x + array_int_size] = (int)(array_float[threadIdx.x]);
	__syncthreads();
	array_int[threadIdx.x]=(int)(shared_mem[s  - 1 - threadIdx.x] * 2);
	array_float[threadIdx.x]=(float)(shared_mem[s - array_float_size- 1 - threadIdx.x] * 2);
}

__host__ void Hanon_Exercise_17()
{

	int array_int_host[4]={1,2,3,4};
	float array_float_host[4]={101.,102.,103.,104.};

	int* array_int_dev=0;
	float* array_float_dev=0;
	hipMalloc((void**)&array_int_dev,4*sizeof(int));
	hipMalloc((void**)&array_float_dev,4*sizeof(float));
	hipMemcpy(array_int_dev,array_int_host,4*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(array_float_dev,array_float_host,4*sizeof(float),hipMemcpyHostToDevice);
	
	/*Your implementation: comment back the following code with the correct specification for shared memory size (by replacing the * with a proper number) */
	Reverse_And_Multiply_Two_Arrays_With_Extern_Shared<<<1,4,4 * sizeof(float) + 4 * sizeof(int)>>>(array_int_dev,4,array_float_dev,4);
	
	hipMemcpy(array_int_host,array_int_dev,4*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(array_float_host,array_float_dev,4*sizeof(float),hipMemcpyDeviceToHost);
	hipFree((void*)array_float_dev);
	hipFree((void*)array_int_dev);

	cout<<"Hanon exercise 17:\n";
	for(int i=0;i<4;i++)cout<<array_int_host[i]<<", ";cout<<endl;
	for(int i=0;i<4;i++)cout<<array_float_host[i]<<", ";cout<<endl;

	out<<"Hanon exercise 17:\n";
	for(int i=0;i<4;i++)out<<array_int_host[i]<<", ";out<<endl;
	for(int i=0;i<4;i++)out<<array_float_host[i]<<", ";out<<endl;
}

////Congratulations! You have finished all your Hanon exercises today!
//////////////////////////////////////////////////////////////////////////

void Hanon_Exercise_Test_Memory()
{
	Hanon_Exercise_12();
	Hanon_Exercise_13();
	Hanon_Exercise_14();
	Hanon_Exercise_15();
	Hanon_Exercise_16();
	Hanon_Exercise_17();
}

int main()
{
	if(name::team=="Team_X"){
		printf("\nPlease specify your team name and team member names in name::team and name::author to start.\n");
		return 0;
	}

	std::string file_name=name::team+"_exercise_memory.dat";
	out.open(file_name.c_str());
	
	if(out.fail()){
		printf("\ncannot open file %s to record results\n",file_name.c_str());
		return 0;
	}

	Hanon_Exercise_Test_Memory();
	return 0;
}
